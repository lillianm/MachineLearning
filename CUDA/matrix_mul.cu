#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#include <math.h>
#include <stdio.h>
#define TILE_WIDTH 16
    namespace cuda
    {
/* update 16*1 result vector at the same time*/
      __device__ void update(float *a, float b, float *c)
      {
        for (int i = 0; i < 16; i++)
          c[i] += a[i * 4] * b;
      }
/*
 * kernel function
 * 16*8 block to calculate block of 16 * 128
 * a boundary issue still exist while using float4 data type
 */    
 __global__ void matrix_mul_kernel (float *a, float *b, float *c, int n)
 {
 /* use shared memory to hold the transpose of a
  * 16 x 16 sub-matrix of 1 x 4 sub-vectors of a
  */
  __shared__ float as[16][65];
 /*  registers for 16 * 1 of c sub-matrix */
  float cr[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
  float4 zero = make_float4(0.0,0.0,0.0,0.0);
/*
 * local variables for each thread to determine which data to fetch and which block to calculate
 */
 int nDiv64 = (n-1)/64+1;
 int sRow = threadIdx.y;
 int sRow4 = sRow*4;
 int sCol = threadIdx.x;
 int tid = sRow*16+sCol;
 int aNext = (16*blockIdx.y+sRow)*n+sCol*4;
 int bNext = 128*blockIdx.x + tid;
 int cNext = 16*blockIdx.y*n + 128*blockIdx.x + tid;
 int nTimes2 = 2*n;
 int nTimes3 = 3*n;
 int nTimes4 = 4*n;
 a += aNext;
 b += bNext;
 c += cNext;
 float4 *a4 = (float4 *)a;
  /*
   * Multiple 16*64 As and 64*16 Bs and then update a 16*1 C 
   */
   for (int i = 0; i < nDiv64; i++)
   {
   //if the next 4 number is not null
    if ( i*64 + sCol *4 < n && blockIdx.y * 16 + sRow < n)
    {
     // if next 4 number is full
      if( i*64 + sCol *4 + 3 < n ){
        *( (float4 *)(&as[sCol][sRow4]) ) = a4[0];
      }
      else
      { 
        // if only 3 numbers 
        if( i*64 + sCol *4 + 2 < n ){
          *( (float4 *)(&as[sCol][sRow4]) ) = make_float4(a4[0].x,a4[0].y,a4[0].z,0.0);
        }
          //if only 2 numbers
        else{ 
          if( i*64 + sCol *4 + 1 < n ){
            *( (float4 *)(&as[sCol][sRow4]) ) = make_float4(a4[0].x,a4[0].y,0.0,0.0);
          }
          else{
            *( (float4 *)(&as[sCol][sRow4]) ) = make_float4(a4[0].x,0.0,0.0,0.0);
          }
        }
      }

    }
    /* if the next 4 numbers are all out of boundary */
    else{
      *( (float4 *)(&as[sCol][sRow4]) ) = zero;
    }
    /* The next 8 rows , same operation as the previous block */
    if ( i*64 + sCol *4 < n && blockIdx.y * 16 + 8 + sRow < n)
    {
      // if next 4 number is not null
      if( i*64 + sCol *4 + 3 < n)
        *( (float4 *)(&as[sCol][sRow4 + 32]) ) = a4[nTimes2];
      else
      {
        // if only 3 numbers
        if( i*64 + sCol *4 + 2 < n ){
          *( (float4 *)(&as[sCol][sRow4 + 32]) ) = make_float4(a4[nTimes2].x,a4[nTimes2].y,a4[nTimes2].z,0.0);
        }
        //if only 2 numbers
        else{
          if( i*64 + sCol *4 + 1 < n ){
            *( (float4 *)(&as[sCol][sRow4 + 32]) ) = make_float4(a4[nTimes2].x,a4[nTimes2].y,0.0,0.0);
          }
          else{
            *( (float4 *)(&as[sCol][sRow4 + 32]) ) = make_float4(a4[nTimes2].x,0.0,0.0,0.0);
          }
        }
      }

    }
    else{
      *( (float4 *)(&as[sCol][sRow4+32]) ) = zero;
    }
    /* wait for read to complete fetching subA */
    __syncthreads(); // wait for read to complete

    /* Begin fetching subB*/
    /* each thread fetch 4*1 at one time and do 16 iteration */
    float br0;
    float br1;
    float br2;
    float br3;
    int boundary_row = blockIdx.x * 128 + threadIdx.y * 16 + threadIdx.x;
    if( boundary_row < n && 64 * i <n)
      br0 = b[0];
    else
      br0 = 0.0;
    if ( boundary_row < n && 64 * i + 1  < n)
      br1 = b[n];
    else
      br1 = 0.0;
    if (boundary_row < n && 64 * i + 2  < n)
      br2 = b[nTimes2];
    else
      br2 = 0.0;
    if (boundary_row < n && 64 * i + 3  < n)
      br3 = b[nTimes3];
    else 
      br3 = 0.0;

    b += nTimes4;

      #pragma unroll
    for (int k = 0; k < 15; k++)
    {
      update (&as[k][0], br0, cr); 
      if( boundary_row < n && 64*i + (k+1)*4   < n )
        br0 = b[0];
      else
        br0 = 0.0;
      update (&as[k][1], br1, cr); 
      if( boundary_row < n && 64*i + (k+1)*4 + 1  < n )
        br1 = b[n];
      else
        br1 = 0.0;
      update (&as[k][2], br2, cr); br2 = b[nTimes2];
      if( boundary_row < n && 64*i + (k+1)*4 + 2 < n )
        br2 = b[nTimes2];
      else
        br2 = 0.0;
      update (&as[k][3], br3, cr); br3 = b[nTimes3];
      if( boundary_row < n && 64*i + (k+1)*4 +  3 < n)
        br3 = b[nTimes3];
      else
        br3 = 0.0;

      b+= nTimes4;
    }
          /* update result value */
    update (&as[15][0], br0, cr);
    update (&as[15][1], br1, cr);
    update (&as[15][2], br2, cr);
    update (&as[15][3], br3, cr);
    a4 += 16;
    __syncthreads(); // wait for computation to complete
  }
  /* return the result value
   * Do not update if out of boundary 
   */
  if(blockIdx.x * 128 + threadIdx.y * 16 + threadIdx.x < n){
    for (int j = 0; j < 16; j++)
    {
      if(blockIdx.y*16 + j < n){
        c[0] = cr[j];
        c += n; 
      }
    }
  }

}
/* Kernel for small matrix*/
__global__ void matrix_mul_kernel_small_matrix(float* A, float* B, float* C, int sq_dimension){

  float CValue = 0;

  int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
  int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;
  int Dim = TILE_WIDTH;
  
    __shared__ float As[TILE_WIDTH][TILE_WIDTH + 1]; // avoid bank conflict
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

    for (int k = 0; k < (TILE_WIDTH + sq_dimension - 1)/TILE_WIDTH; k++) {

     if (k*TILE_WIDTH + threadIdx.x < sq_dimension && Row < sq_dimension)   
      As[threadIdx.y][threadIdx.x] = A[Row*sq_dimension + k*TILE_WIDTH + threadIdx.x];
    else                                                   
      As[threadIdx.y][threadIdx.x] = 0.0;

    if (k*TILE_WIDTH + threadIdx.y < sq_dimension && Col < sq_dimension)   
      Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_WIDTH + threadIdx.y)*sq_dimension + Col];
    else                                                   
      Bs[threadIdx.y][threadIdx.x] = 0.0;

    __syncthreads();
    for (int n = 0; n < Dim; ++n) 
      CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];


    __syncthreads();
  }

  if (Row < sq_dimension && Col < sq_dimension) 
    C[((blockIdx.y * blockDim.y + threadIdx.y)*sq_dimension)+(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;
}
void 
matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
{
  int size = sq_dimension * sq_dimension * sizeof(float);
  float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;

    /***************************************************
  1st Part: Allocation of memory on device memory  
    ****************************************************/

    /* copy sq_matrix_1 and sq_matrix_2 to device memory */
  hipMalloc((void**) &sq_matrix_1_d, size);
  hipMemcpy(sq_matrix_1_d, sq_matrix_1, size, hipMemcpyHostToDevice);
  hipMalloc((void**) &sq_matrix_2_d, size);
  hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);

    /*allocate sq_matrix_result on host */
  hipMalloc((void**) &sq_matrix_result_d, size);

    /***************************************************
   2nd Part: Inovke kernel 
    ****************************************************/
   if(sq_dimension % 4 !=0){
    /* All matrix valid*/
    dim3 dimBlock(16, 16);
    dim3 dimGrid((sq_dimension-1)/16 + 1,(sq_dimension -1)/16+1);
    matrix_mul_kernel_small_matrix<<<dimGrid, dimBlock,dimBlock.x * dimBlock.y*sizeof(float)>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);

  }
  else{
    /* Should be able to work with any input dimension, but does not work with odd numbers, still debugging */
   dim3 dimBlock(16, 8);
   dim3 dimGrid((sq_dimension-1)/128 + 1,(sq_dimension -1)/16+1);
   matrix_mul_kernel<<<dimGrid, dimBlock,dimBlock.x * dimBlock.y*sizeof(float)>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
 }


    /***************************************************
   3rd Part: Transfer result from device to host 
    ****************************************************/
   hipMemcpy(sq_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);
   hipFree(sq_matrix_1_d);
   hipFree(sq_matrix_2_d);
   hipFree(sq_matrix_result_d);
 }  
} // namespace cuda
